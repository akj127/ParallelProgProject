#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <opencv2/opencv.hpp>
#include <iostream>
#include <cmath>

// Helper macros for CUDA error checking
#define CUDA_CHECK(call)                                                      \
    {                                                                         \
        hipError_t err = call;                                               \
        if (err != hipSuccess) {                                             \
            std::cerr << "CUDA error in " << __FILE__ << ":" << __LINE__      \
                      << " - " << hipGetErrorString(err) << std::endl;       \
            exit(EXIT_FAILURE);                                               \
        }                                                                     \
    }

// Host Function Declarations
void loadAndPreprocessImage(const char* filename, float** h_image, int* width, int* height);
void applyGaussianBlur(float* d_image, float* d_blurred, int width, int height);
void computeGradients(float* d_blurred, float* d_gradient, float* d_direction, int width, int height);
void performNonMaxSuppression(float* d_gradient, float* d_direction, float* d_edges, int width, int height);
void applyDoubleThresholding(float* d_edges, float highThreshold, float lowThreshold, int width, int height);
void edgeTrackingByHysteresis(float* d_edges, int width, int height);
void saveOutputImage(const char* filename, float* d_edges, int width, int height);

// CUDA Kernel Declarations
__global__ void GaussianBlurKernel(float* d_image, float* d_blurred, int width, int height);
__global__ void SobelKernel(float* d_blurred, float* d_gradient, float* d_direction, int width, int height);
__global__ void NonMaxSuppressionKernel(float* d_gradient, float* d_direction, float* d_edges, int width, int height);
__global__ void DoubleThresholdKernel(float* d_edges, float highThreshold, float lowThreshold, int width, int height);
__global__ void HysteresisKernel(float* d_edges, int width, int height);

int main(int argc, char** argv) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <image_file>" << std::endl;
        return -1;
    }

    const char* inputImage = argv[1];
    const char* outputImage = "output_edges.png";

    // Host and Device variables
    float* h_image = nullptr;
    int width, height;

    float *d_image, *d_blurred, *d_gradient, *d_direction, *d_edges;

    // Load and preprocess the input image
    loadAndPreprocessImage(inputImage, &h_image, &width, &height);

    // Allocate device memory
    CUDA_CHECK(hipMalloc(&d_image, width * height * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_blurred, width * height * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_gradient, width * height * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_direction, width * height * sizeof(float)));
    CUDA_CHECK(hipMalloc(&d_edges, width * height * sizeof(float)));

    // Copy input image to device memory
    CUDA_CHECK(hipMemcpy(d_image, h_image, width * height * sizeof(float), hipMemcpyHostToDevice));

    // Perform each stage of Canny Edge Detection
    applyGaussianBlur(d_image, d_blurred, width, height);
    computeGradients(d_blurred, d_gradient, d_direction, width, height);
    performNonMaxSuppression(d_gradient, d_direction, d_edges, width, height);
    applyDoubleThresholding(d_edges, 0.2f, 0.1f, width, height);
    edgeTrackingByHysteresis(d_edges, width, height);

    // Save the final output image
    saveOutputImage(outputImage, d_edges, width, height);

    // Free device memory
    CUDA_CHECK(hipFree(d_image));
    CUDA_CHECK(hipFree(d_blurred));
    CUDA_CHECK(hipFree(d_gradient));
    CUDA_CHECK(hipFree(d_direction));
    CUDA_CHECK(hipFree(d_edges));

    // Free host memory
    free(h_image);

    return 0;
}

void loadAndPreprocessImage(const char* filename, float** h_image, int* width, int* height) {
    cv::Mat img = cv::imread(filename, cv::IMREAD_GRAYSCALE);
    if (img.empty()) {
        std::cerr << "Error: Could not load image." << std::endl;
        exit(EXIT_FAILURE);
    }

    *width = img.cols;
    *height = img.rows;
    *h_image = (float*)malloc((*width) * (*height) * sizeof(float));

    for (int i = 0; i < img.rows; ++i) {
        for (int j = 0; j < img.cols; ++j) {
            (*h_image)[i * (*width) + j] = img.at<uchar>(i, j) / 255.0f;
        }
    }
}

void saveOutputImage(const char* filename, float* d_edges, int width, int height) {
    float* h_edges = (float*)malloc(width * height * sizeof(float));
    CUDA_CHECK(hipMemcpy(h_edges, d_edges, width * height * sizeof(float), hipMemcpyDeviceToHost));

    cv::Mat output(height, width, CV_32F, h_edges);
    output.convertTo(output, CV_8U, 255.0);
    cv::imwrite(filename, output);

    free(h_edges);
}

void applyGaussianBlur(float* d_image, float* d_blurred, int width, int height) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    GaussianBlurKernel<<<gridSize, blockSize>>>(d_image, d_blurred, width, height);
    CUDA_CHECK(hipDeviceSynchronize());
}

void computeGradients(float* d_blurred, float* d_gradient, float* d_direction, int width, int height) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    SobelKernel<<<gridSize, blockSize>>>(d_blurred, d_gradient, d_direction, width, height);
    CUDA_CHECK(hipDeviceSynchronize());
}

void performNonMaxSuppression(float* d_gradient, float* d_direction, float* d_edges, int width, int height) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    NonMaxSuppressionKernel<<<gridSize, blockSize>>>(d_gradient, d_direction, d_edges, width, height);
    CUDA_CHECK(hipDeviceSynchronize());
}

void applyDoubleThresholding(float* d_edges, float highThreshold, float lowThreshold, int width, int height) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    DoubleThresholdKernel<<<gridSize, blockSize>>>(d_edges, highThreshold, lowThreshold, width, height);
    CUDA_CHECK(hipDeviceSynchronize());
}

void edgeTrackingByHysteresis(float* d_edges, int width, int height) {
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    HysteresisKernel<<<gridSize, blockSize>>>(d_edges, width, height);
    CUDA_CHECK(hipDeviceSynchronize());
}

__global__ void GaussianBlurKernel(float* d_image, float* d_blurred, int width, int height) {
    // Implement Gaussian blur logic
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    const float kernel[5][5] = {
        {1, 4, 7, 4, 1},
        {4, 16, 26, 16, 4},
        {7, 26, 41, 26, 7},
        {4, 16, 26, 16, 4},
        {1, 4, 7, 4, 1}
    };
    float sum = 0;

    for (int ky = -2; ky <= 2; ++ky) {
        for (int kx = -2; kx <= 2; ++kx) {
            int neighborX = min(max(x + kx, 0), width - 1);
            int neighborY = min(max(y + ky, 0), height - 1);
            sum += kernel[ky + 2][kx + 2] * d_image[neighborY * width + neighborX];
        }
    }
    d_blurred[y * width + x] = sum / 255.0; 
    
}

__global__ void SobelKernel(float* d_blurred, float* d_gradient, float* d_direction, int width, int height) {
    // Implement gradient computation logic
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    const int Gx[3][3] = {
        {-1, 0, 1},
        {-2, 0, 2},
        {-1, 0, 1}
    };
    const int Gy[3][3] = {
        { 1,  2,  1},
        { 0,  0,  0},
        {-1, -2, -1}
    };
    float sumX = 0.0f, sumY = 0.0f;
    for (int ky = -1; ky <= 1; ++ky) {
        for (int kx = -1; kx <= 1; ++kx) {
            int neighborX = min(max(x + kx, 0), width - 1);
            int neighborY = min(max(y + ky, 0), height - 1);
            float pixel = d_blurred[neighborY * width + neighborX];
            sumX += Gx[ky + 1][kx + 1] * pixel;
            sumY += Gy[ky + 1][kx + 1] * pixel;
        }
    }
    d_gradient[y * width + x] = sqrtf(sumX * sumX + sumY * sumY);
    d_direction[y * width + x] = atan2f(sumY, sumX);
}

__global__ void NonMaxSuppressionKernel(float* d_gradient, float* d_direction, float* d_edges, int width, int height) {
    // Implement non-maximum suppression logic
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;
    float direction = d_direction[y * width + x];
    float magnitude = d_gradient[y * width + x];
    int dx1 = 0, dy1 = 0, dx2 = 0, dy2 = 0;

    if ((direction >= -M_PI / 8 && direction < M_PI / 8) || (direction >= 7 * M_PI / 8 || direction < -7 * M_PI / 8)) {
        dx1 = 1; dy1 = 0; dx2 = -1; dy2 = 0;
    } else if ((direction >= M_PI / 8 && direction < 3 * M_PI / 8) || (direction >= -7 * M_PI / 8 && direction < -5 * M_PI / 8)) {
        dx1 = 1; dy1 = 1; dx2 = -1; dy2 = -1;
    } else if ((direction >= 3 * M_PI / 8 && direction < 5 * M_PI / 8) || (direction >= -5 * M_PI / 8 && direction < -3 * M_PI / 8)) {
        dx1 = 0; dy1 = 1; dx2 = 0; dy2 = -1;
    } else {
        dx1 = -1; dy1 = 1; dx2 = 1; dy2 = -1;
    }

    int neighbor1X = min(max(x + dx1, 0), width - 1);
    int neighbor1Y = min(max(y + dy1, 0), height - 1);
    int neighbor2X = min(max(x + dx2, 0), width - 1);
    int neighbor2Y = min(max(y + dy2, 0), height - 1);

    if (magnitude >= d_gradient[neighbor1Y * width + neighbor1X] &&
        magnitude >= d_gradient[neighbor2Y * width + neighbor2X]) {
        d_edges[y * width + x] = magnitude;
    } else {
        d_edges[y * width + x] = 0.0f;
    }
}

__global__ void DoubleThresholdKernel(float* d_edges, float highThreshold, float lowThreshold, int width, int height) {
    // Implement double thresholding logic
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    float value = d_edges[y * width + x];
    if (value >= highThreshold) {
        d_edges[y * width + x] = 1.0f;
    } else if (value >= lowThreshold) {
        d_edges[y * width + x] = 0.5f;
    } else {
        d_edges[y * width + x] = 0.0f;
    }
}

__global__ void HysteresisKernel(float* d_edges, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height) return;

    if (d_edges[y * width + x] != 0.5f) return; // Process only weak edges

    bool connectedToStrong = false;
    for (int dy = -1; dy <= 1; ++dy) {
        for (int dx = -1; dx <= 1; ++dx) {
            if (dx == 0 && dy == 0) continue;

            int neighborX = min(max(x + dx, 0), width - 1);
            int neighborY = min(max(y + dy, 0), height - 1);
            if (d_edges[neighborY * width + neighborX] == 1.0f) {
                connectedToStrong = true;
                break;
            }
        }
        if (connectedToStrong) break;
    }

    d_edges[y * width + x] = connectedToStrong ? 1.0f : 0.0f;
}

